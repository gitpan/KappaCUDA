#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixMul.h"

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) cutilBankChecker(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) cutilBankChecker(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif

__constant__ float constData[8192];

// declare texture reference for 1D float texture
texture<float, 1, hipReadModeElementType> tex_1D;

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex_2D;

// declare texture reference for 3D float texture
texture<float, 3, hipReadModeElementType> tex_3D;

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
matrixMul( float* C, float* A, float* B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        AS(ty, tx) = A[a + wA * ty + tx];
        BS(ty, tx) = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += AS(ty, k) * BS(k, tx);

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

////////////////////////////////////////////////////////////////////////////////
//! Fetch from texture lookup
//! @param g_odata output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
fetchTexOneD( float* g_odata, int width) 
{
    // calculate normalized texture coordinates
    unsigned int x = threadIdx.x;

    // read from texture and write to global memory
    g_odata[x] = tex1Dfetch(tex_1D, x);
}

////////////////////////////////////////////////////////////////////////////////
//! Get from texture lookup
//! @param g_odata output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
getTexOneD( float* g_odata, int width) 
{
    // calculate normalized texture coordinates
    unsigned int x = threadIdx.x;

    float u = ((float)x + 0.5f)/ (float) width;

    // read from texture and write to global memory
    g_odata[x] = tex1D(tex_1D, u);
}

////////////////////////////////////////////////////////////////////////////////
//! Get from texture lookup
//! @param g_odata output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
getTexTwoD( float* g_odata, int width, int height) 
{
    // calculate normalized texture coordinates
    unsigned int x = threadIdx.x;
    unsigned int y = threadIdx.y;

    float u = ((float)x + 0.25f)/ (float) width;
    float v = ((float)y + 0.25f)/ (float) height;

    // read from texture and write to global memory
    g_odata[y*width + x] = tex2D(tex_2D, u, v);
}

////////////////////////////////////////////////////////////////////////////////
//! Get from texture lookup
//! @param g_odata output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
getTexThreeD( float* g_odata, int width, int height, int depth) 
{
    // calculate normalized texture coordinates
    unsigned int x = threadIdx.x;
    unsigned int y = threadIdx.y;
    unsigned int z = threadIdx.z;

    float u = ((float)x + 0.75f)/ (float) width;
    float v = ((float)y + 0.75f)/ (float) height;
    float w = ((float)z + 0.75f)/ (float) depth;

    // read from texture and write to global memory
    g_odata[(height*z + y)*width + x] = tex3D(tex_3D, u, v, w);
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
